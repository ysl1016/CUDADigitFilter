#include "hip/hip_runtime.h"
#include "filters.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <arpa/inet.h>  // For ntohl function
#include <iostream>

#define BLOCK_SIZE 16

__global__ void sobelFilterKernel(const unsigned char *input, unsigned char *output, int width, int height) {
    // Shared memory for the block
    __shared__ unsigned char sharedMem[BLOCK_SIZE + 2][BLOCK_SIZE + 2];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int threadX = threadIdx.x + 1; // +1 for halo
    int threadY = threadIdx.y + 1;

    // Load data into shared memory
    if (x < width && y < height) {
        sharedMem[threadY][threadX] = input[y * width + x];

        // Load halo pixels
        if (threadIdx.x == 0 && x > 0)
            sharedMem[threadY][0] = input[y * width + (x - 1)];
        if (threadIdx.x == blockDim.x - 1 && x < width - 1)
            sharedMem[threadY][threadX + 1] = input[y * width + (x + 1)];
        if (threadIdx.y == 0 && y > 0)
            sharedMem[0][threadX] = input[(y - 1) * width + x];
        if (threadIdx.y == blockDim.y - 1 && y < height - 1)
            sharedMem[threadY + 1][threadX] = input[(y + 1) * width + x];

        if (threadIdx.x == 0 && threadIdx.y == 0 && x > 0 && y > 0)
            sharedMem[0][0] = input[(y - 1) * width + (x - 1)];
        if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0 && x < width - 1 && y > 0)
            sharedMem[0][threadX + 1] = input[(y - 1) * width + (x + 1)];
        if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1 && x > 0 && y < height - 1)
            sharedMem[threadY + 1][0] = input[(y + 1) * width + (x - 1)];
        if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1 && x < width - 1 && y < height - 1)
            sharedMem[threadY + 1][threadX + 1] = input[(y + 1) * width + (x + 1)];
    }
    __syncthreads();

    // Apply Sobel filter
    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        int Gx =
            -1 * sharedMem[threadY - 1][threadX - 1] + 1 * sharedMem[threadY - 1][threadX + 1] +
            -2 * sharedMem[threadY][threadX - 1] + 2 * sharedMem[threadY][threadX + 1] +
            -1 * sharedMem[threadY + 1][threadX - 1] + 1 * sharedMem[threadY + 1][threadX + 1];

        int Gy =
            -1 * sharedMem[threadY - 1][threadX - 1] + -2 * sharedMem[threadY - 1][threadX] + -1 * sharedMem[threadY - 1][threadX + 1] +
             1 * sharedMem[threadY + 1][threadX - 1] +  2 * sharedMem[threadY + 1][threadX] +  1 * sharedMem[threadY + 1][threadX + 1];

        int magnitude = sqrtf(Gx * Gx + Gy * Gy);
        output[y * width + x] = min(max(magnitude, 0), 255);
    }
}

__global__ void gaussianBlur(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        if (x < 2 || x >= width-2 || y < 2 || y >= height-2) {
            output[y * width + x] = input[y * width + x];
            return;
        }

        float gaussian[5][5] = {
            {1, 4, 6, 4, 1},
            {4, 16, 24, 16, 4},
            {6, 24, 36, 24, 6},
            {4, 16, 24, 16, 4},
            {1, 4, 6, 4, 1}
        };
        
        float sum = 0;
        float weightSum = 0;
        
        for(int i = -2; i <= 2; i++) {
            for(int j = -2; j <= 2; j++) {
                float weight = gaussian[i+2][j+2];
                sum += input[(y+i)*width + (x+j)] * weight;
                weightSum += weight;
            }
        }
        
        output[y * width + x] = static_cast<unsigned char>(sum / weightSum);
    }
}

__global__ void sharpenFilter(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        if (x == 0 || x == width-1 || y == 0 || y == height-1) {
            output[y * width + x] = input[y * width + x];
            return;
        }

        int kernel[3][3] = {
            {0, -1, 0},
            {-1, 5, -1},
            {0, -1, 0}
        };
        
        int sum = 0;
        for(int i = -1; i <= 1; i++) {
            for(int j = -1; j <= 1; j++) {
                sum += input[(y+i)*width + (x+j)] * kernel[i+1][j+1];
            }
        }
        
        // Using ternary operators instead of std::min/max
        sum = sum < 0 ? 0 : (sum > 255 ? 255 : sum);
        output[y * width + x] = static_cast<unsigned char>(sum);
    }
}

void applyFilter(unsigned char* d_input, unsigned char* d_output, 
                int width, int height, FilterType filter_type) {
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    // CUDA 장치 초기화
    hipDeviceReset();

    // 각 이미지에 대해 필터 적용
    switch(filter_type) {
        case FilterType::SOBEL:
            sobelFilter<<<gridSize, blockSize>>>(d_input, d_output, width, height);
            break;
        case FilterType::GAUSSIAN:
            gaussianBlur<<<gridSize, blockSize>>>(d_input, d_output, width, height);
            break;
        case FilterType::SHARPEN:
            sharpenFilter<<<gridSize, blockSize>>>(d_input, d_output, width, height);
            break;
    }
    
    // CUDA 오류 체크 추가
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    
    // 커널 실행 후 동기화
    hipDeviceSynchronize();
}
