#include "hip/hip_runtime.h"
#include "filters.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <arpa/inet.h>  // For ntohl function

__global__ void sobelFilter(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        // Skip border pixels
        if (x == 0 || x == width-1 || y == 0 || y == height-1) {
            output[y * width + x] = input[y * width + x];
            return;
        }

        // Sobel kernels
        int Gx = input[(y-1)*width + (x+1)] + 2*input[y*width + (x+1)] + input[(y+1)*width + (x+1)]
                 - input[(y-1)*width + (x-1)] - 2*input[y*width + (x-1)] - input[(y+1)*width + (x-1)];

        int Gy = input[(y+1)*width + (x-1)] + 2*input[(y+1)*width + x] + input[(y+1)*width + (x+1)]
                 - input[(y-1)*width + (x-1)] - 2*input[(y-1)*width + x] - input[(y-1)*width + (x+1)];
                 
        float val = sqrtf((float)(Gx*Gx + Gy*Gy));
        output[y * width + x] = (unsigned char)(val > 255 ? 255 : val);
    }
}

__global__ void gaussianBlur(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        if (x < 2 || x >= width-2 || y < 2 || y >= height-2) {
            output[y * width + x] = input[y * width + x];
            return;
        }

        float gaussian[5][5] = {
            {1, 4, 6, 4, 1},
            {4, 16, 24, 16, 4},
            {6, 24, 36, 24, 6},
            {4, 16, 24, 16, 4},
            {1, 4, 6, 4, 1}
        };
        
        float sum = 0;
        float weightSum = 0;
        
        for(int i = -2; i <= 2; i++) {
            for(int j = -2; j <= 2; j++) {
                float weight = gaussian[i+2][j+2];
                sum += input[(y+i)*width + (x+j)] * weight;
                weightSum += weight;
            }
        }
        
        output[y * width + x] = static_cast<unsigned char>(sum / weightSum);
    }
}

__global__ void sharpenFilter(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        if (x == 0 || x == width-1 || y == 0 || y == height-1) {
            output[y * width + x] = input[y * width + x];
            return;
        }

        int kernel[3][3] = {
            {0, -1, 0},
            {-1, 5, -1},
            {0, -1, 0}
        };
        
        int sum = 0;
        for(int i = -1; i <= 1; i++) {
            for(int j = -1; j <= 1; j++) {
                sum += input[(y+i)*width + (x+j)] * kernel[i+1][j+1];
            }
        }
        
        // Using ternary operators instead of std::min/max
        sum = sum < 0 ? 0 : (sum > 255 ? 255 : sum);
        output[y * width + x] = static_cast<unsigned char>(sum);
    }
}

void applyFilter(unsigned char* d_input, unsigned char* d_output, 
                int width, int height, FilterType filter_type) {
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    // CUDA 장치 초기화
    hipDeviceReset();

    // 각 이미지에 대해 필터 적용
    switch(filter_type) {
        case FilterType::SOBEL:
            sobelFilter<<<gridSize, blockSize>>>(d_input, d_output, width, height);
            break;
        case FilterType::GAUSSIAN:
            gaussianBlur<<<gridSize, blockSize>>>(d_input, d_output, width, height);
            break;
        case FilterType::SHARPEN:
            sharpenFilter<<<gridSize, blockSize>>>(d_input, d_output, width, height);
            break;
    }
    
    // CUDA 오류 체크 추가
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    
    // 커널 실행 후 동기화
    hipDeviceSynchronize();
}
